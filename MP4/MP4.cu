#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <time.h>
#include <>

#define BLOCK_WIDTH 16
#define TILE_WIDTH 2

__global__ void matrixMultiplication(float *M, float *N, float *P, int matrixSize) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	float sum = 0.0f;
	if (row < matrixSize && col < matrixSize) {
		for (int k = 0; k < matrixSize; k++) {
			sum += M[row * matrixSize + k] * N[k * matrixSize + col];
		}
		P[row * matrixSize + col] = sum;
	}
}

// Function to perform matrix multiplication on CPU
void matMulCPU(float *M, float *N, float *P, int matrixSize) {
	for (int i = 0; i < matrixSize; i++) {
		for (int j = 0; j < matrixSize; j++) {
			float Pvalue = 0;
			for (int k = 0; k < matrixSize; k++) {
				Pvalue += M[j * matrixSize + k] * N[k * matrixSize + i];
			}
			P[j * matrixSize + i] = Pvalue;
		}
	}
}


__global__ void tiled_matrix_multiply(float *A, float *B, float *C, int n)
{
	__shared__ float tileA[TILE_WIDTH][TILE_WIDTH];
	__shared__ float tileB[TILE_WIDTH][TILE_WIDTH];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bx = blockIdx.x;
	int by = blockIdx.y;

	int row = by * TILE_WIDTH + ty;
	int col = bx * TILE_WIDTH + tx;

	float sum = 0;

	for (int i = 0; i < n / TILE_WIDTH; i++) {
		tileA[ty][tx] = A[row * n + (i * TILE_WIDTH + tx)];
		tileB[ty][tx] = B[(i * TILE_WIDTH + ty) * n + col];
		__syncthreads();

		for (int j = 0; j < TILE_WIDTH; j++) {
			sum += tileA[ty][j] * tileB[j][tx];
		}
		__syncthreads();
	}

	C[row * n + col] = sum;
}

int main()
{
	// Sizes of input matrices to test
	int sizeOfBlock = 1;
	int sizes[] = { 125, 250, 500, 1000, 2000 };
	int blockSizes[] = { 2,4,10,20,25 };
	//sizeOfBlock = blockSizes[1];
	//printf("Tile width of: %d\n", TILE_WIDTH);

	// Loop over matrix sizes
	for (int x = 0; x < 5; x++) {
		sizeOfBlock = blockSizes[x];
		printf("Block width of: %d\n", sizeOfBlock);
		for (int i = 0; i < 5; i++)
		{
			int size = sizes[i];
			//sizeOfBlock = blockSizes[x];
			printf("Matrix size is %d by %d\n\n", size, size);
			size_t hostSize = size * size * sizeof(float);

			float gpu_time1 = 0.0f;
			float gpu_time2 = 0.0f;

			// Allocate memory for input matrices on host
			float* h_M = (float*)malloc(hostSize);
			float* h_N = (float*)malloc(hostSize);
			float* h_C_GPU = (float*)malloc(hostSize);
			float* h_C_CPU = (float*)malloc(hostSize);


			srand(time(NULL));
			for (int i = 0; i < size * size; i++) {
				h_M[i] = (float)rand() / RAND_MAX;
				h_N[i] = (float)rand() / RAND_MAX;
			}

			// Allocate memory for input matrices on device
			float *d_M, *d_N, *d_C;
			hipMalloc(&d_M, hostSize);
			hipMalloc(&d_N, hostSize);
			hipMalloc(&d_C, hostSize);

			// Create events to measure time
			hipEvent_t start1, stop1, start2, stop2;
			hipEventCreate(&start1);
			hipEventCreate(&stop1);
			hipEventCreate(&start2);
			hipEventCreate(&stop2);

			//Host Multiplication
			hipEventRecord(start1, 0);
			printf("test");
			matMulCPU(h_M, h_N, h_C_CPU, size);
			hipEventRecord(stop1, 0);

			hipEventElapsedTime(&gpu_time1, start1, stop1);
			printf("Host Multiplication time: %0.2f\n", gpu_time1);

			// Copy input matrices from host to device and measure time
			//hipEventRecord(start1);
			hipMemcpy(d_M, h_M, hostSize, hipMemcpyHostToDevice);
			hipMemcpy(d_N, h_N, hostSize, hipMemcpyHostToDevice);
			// hipEventRecord(stop1);
			// hipEventSynchronize(stop1);
			// float transfer_time = 0;
			// hipEventElapsedTime(&transfer_time, start1, stop1);
			// printf("Matrix size %d x %d: Host to device transfer time = %f ms\n", size, size, transfer_time);

			int NumBlocks = size / sizeOfBlock;
			if (size % sizeOfBlock) NumBlocks++;
			//dim3 numberOfBlocks(NumBlocks, NumBlocks);
			//dim3 threadsPerBlock(sizeOfBlock, sizeOfBlock);

			dim3 threadsPerBlock(TILE_WIDTH, TILE_WIDTH);
			dim3 numberOfBlocks(size / TILE_WIDTH, size / TILE_WIDTH);


			// //Part 2 ---------------------------------------------------------------------
			hipEventRecord(start2, 0);
			matrixMultiplication << < numberOfBlocks, threadsPerBlock >> >(d_M, d_N, d_C, size);
			hipEventRecord(stop2, 0);
			hipEventSynchronize(stop2);
			hipEventElapsedTime(&gpu_time2, start2, stop2);
			hipMemcpy(h_C_GPU, d_C, hostSize, hipMemcpyDeviceToHost);
			printf("Normal Multiplication time: %0.2f\n", gpu_time2);

			//for (int i = 0; i < size * size; i++) {
			//	if (abs(h_C_CPU[i] - h_C_GPU[i]) > 0.1) {
			//		printf("Test FAILED\n");
			//		break;
			//	}
			//}
			//printf("Test PASSED\n\n");

			//-----------------------------------------------------------------------------

			// Copy input matrices from device to host and measure time
			// hipEventRecord(start1);
			hipMemcpy(h_M, d_M, hostSize, hipMemcpyDeviceToHost);
			hipMemcpy(h_N, d_N, hostSize, hipMemcpyDeviceToHost);
			// hipEventRecord(stop1);
			// hipEventSynchronize(stop1);
			// transfer_time = 0;
			// hipEventElapsedTime(&transfer_time, start1, stop1);
			// printf("Matrix size %d x %d: Device to host transfer time = %f ms\n", size, size, transfer_time);

			// Free memory
			hipHostFree(h_M);
			hipHostFree(h_N);
			hipHostFree(h_C_CPU);
			hipHostFree(h_C_GPU);
			hipFree(d_M);
			hipFree(d_N);
			hipFree(d_C);

		}
		printf("\n");
	}
	return 0;
}