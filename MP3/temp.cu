#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <hiprand/hiprand.h>
#include <time.h>


#define BLOCK_WIDTH 16

__global__ void matrixMultiplication(float *M, float *N, float *P, int matrixSize) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	float sum = 0.0f;
	if (row < matrixSize && col < matrixSize) {
		for (int k = 0; k < matrixSize; k++) {
			sum += M[row * matrixSize + k] * N[k * matrixSize + col];
		}
		P[row * matrixSize + col] = sum;
	}
}

// Function to perform matrix multiplication on CPU
void matMulCPU(float *M, float *N, float *P, int matrixSize){
	for (int i = 0; i < matrixSize; i++) {
		for (int j = 0; j < matrixSize; j++) {
			float Pvalue = 0;
			for (int k = 0; k < matrixSize; k++) {
				Pvalue += M[j * matrixSize + k] * N[k * matrixSize + i];
			}
			P[j * matrixSize + i] = Pvalue;
		}
	}
}


int main()
{
	// Sizes of input matrices to test
	int sizes[] = { 125, 250, 500, 1000, 2000 };
	int num_sizes = sizeof(sizes) / sizeof(int);

	// Loop over matrix sizes
	for (int i = 0; i < num_sizes; i++)
	{
		int size = sizes[i];
		int num_elements = size * size;

		// Allocate memory for input matrices on host
		float *h_M, *h_N;
		hipHostMalloc(&h_M, num_elements * sizeof(float), hipHostMallocDefault);
		hipHostMalloc(&h_N, num_elements * sizeof(float), hipHostMallocDefault);

		srand(time(NULL));
		for (int i = 0; i < size * size; i++) {
			h_M[i] = (float)rand() / RAND_MAX;
			h_N[i] = (float)rand() / RAND_MAX;
		}

		// Allocate memory for input matrices on device
		float *d_M, *d_N;
		hipMalloc(&d_M, num_elements * sizeof(float));
		hipMalloc(&d_N, num_elements * sizeof(float));

		// Create events to measure time
		hipEvent_t start1, stop1, start2, stop2;
		hipEventCreate(&start1);
		hipEventCreate(&stop1);
		hipEventCreate(&start2);
		hipEventCreate(&stop2);

		// Copy input matrices from host to device and measure time
		hipEventRecord(start1);
		hipMemcpy(d_M, h_M, num_elements * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_N, h_N, num_elements * sizeof(float), hipMemcpyHostToDevice);
		hipEventRecord(stop1);
		hipEventSynchronize(stop1);
		float transfer_time = 0;
		hipEventElapsedTime(&transfer_time, start1, stop1);
		printf("Matrix size %d x %d: Host to device transfer time = %f ms\n", size, size, transfer_time);

		// Copy input matrices from device to host and measure time
		hipEventRecord(start1);
		hipMemcpy(h_M, d_M, num_elements * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(h_N, d_N, num_elements * sizeof(float), hipMemcpyDeviceToHost);
		hipEventRecord(stop1);
		hipEventSynchronize(stop1);
		transfer_time = 0;
		hipEventElapsedTime(&transfer_time, start1, stop1);
		printf("Matrix size %d x %d: Device to host transfer time = %f ms\n", size, size, transfer_time);

		// Free memory
		hipHostFree(h_M);
		hipHostFree(h_N);
		hipFree(d_M);
		hipFree(d_N);
	}

	return 0;
}
