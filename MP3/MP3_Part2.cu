#include "hip/hip_runtime.h"
//Luka Gobovic
//20215231
//MP3 Part 2
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <time.h>
#include <>

__global__ void matrixMultiplication(float *M, float *N, float *P, int matrixSize) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < matrixSize && col < matrixSize) {
		float pValue = 0;
		for (int k = 0; k < matrixSize; ++k) {
			pValue += M[row * matrixSize + k] * N[k * matrixSize + col];
		}
		P[row * matrixSize + col] = pValue;
	}
}

// Function to perform matrix multiplication on CPU
void matMulCPU(float *M, float *N, float *P, int matrixSize) {
	for (int i = 0; i < matrixSize; i++) {
		for (int j = 0; j < matrixSize; j++) {
			float Pvalue = 0;
			for (int k = 0; k < matrixSize; k++) {
				Pvalue += M[j * matrixSize + k] * N[k * matrixSize + i];
			}
			P[j * matrixSize + i] = Pvalue;
		}
	}
}


int main()
{
	// Sizes of input matrices to test
	int sizes[] = { 125, 250, 500, 1000, 2000 };

	for (int i = 0; i < 5; i++)
	{
		int size = sizes[i];
		printf("Matrix size is %d by %d\n\n", size, size);
		size_t hostSize = size * size * sizeof(float);

		float gpu_time1 = 0.0f;
		float gpu_time2 = 0.0f;

		// Allocate memory for input matrices on host
		float* h_M = (float*)malloc(hostSize);
		float* h_N = (float*)malloc(hostSize);
		float* h_C_GPU = (float*)malloc(hostSize);
		float* h_C_CPU = (float*)malloc(hostSize);

		srand(time(NULL));
		for (int i = 0; i < size * size; i++) {
			h_M[i] = (float)rand() / RAND_MAX;
			h_N[i] = (float)rand() / RAND_MAX;
		}

		// Allocate memory for input matrices on device
		float *d_M, *d_N, *d_C;
		hipMalloc(&d_M, hostSize);
		hipMalloc(&d_N, hostSize);
		hipMalloc(&d_C, hostSize);

		// Create events to measure time
		hipEvent_t start1, stop1, start2, stop2;
		hipEventCreate(&start1);
		hipEventCreate(&stop1);
		hipEventCreate(&start2);
		hipEventCreate(&stop2);

		//Host Multiplication
		hipEventRecord(start1, 0);
		matMulCPU(h_M, h_N, h_C_CPU, size);
		hipEventRecord(stop1, 0);

		hipEventElapsedTime(&gpu_time1, start1, stop1);
		printf("Host Multiplication time: %0.2f\n", gpu_time1);

		hipMemcpy(d_M, h_M, hostSize, hipMemcpyHostToDevice);
		hipMemcpy(d_N, h_N, hostSize, hipMemcpyHostToDevice);

        //One thread per block and one total block
        dim3 threadsPerBlock(1,1,1);
		dim3 numberOfBlocks(ceil(size / (float)threadsPerBlock.x), ceil(size / (float)threadsPerBlock.y), 1);

		// //Part 2 ---------------------------------------------------------------------
		hipEventRecord(start2, 0);
		matrixMultiplication << <numberOfBlocks, threadsPerBlock >> >(d_M, d_N, d_C, size);
		hipEventRecord(stop2, 0);
		hipEventSynchronize(stop2);
		hipEventElapsedTime(&gpu_time2, start2, stop2);
		hipMemcpy(h_C_GPU, d_C, hostSize, hipMemcpyDeviceToHost);
		printf("Normal Multiplication time: %0.2f\n", gpu_time2);

		for (int i = 0; i < size * size; i++) {
			if (abs(h_C_CPU[i] - h_C_GPU[i]) > 0.00001) {
				printf("Test FAILED\n");
				break;
			}
		}
		printf("Test PASSED\n\n");

		// Free memory
		hipHostFree(h_M);
		hipHostFree(h_N);
		hipHostFree(h_C_CPU);
		hipHostFree(h_C_GPU);
		hipFree(d_M);
		hipFree(d_N);
		hipFree(d_C);
	}
	return 0;
}
