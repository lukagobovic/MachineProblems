#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <time.h>
#include <>

#define BLOCK_WIDTH 16

__global__ void matrixMultiplication(float *M, float *N, float *P, int matrixSize) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	float sum = 0.0f;
	if (row < matrixSize && col < matrixSize) {
		for (int k = 0; k < matrixSize; k++) {
			sum += M[row * matrixSize + k] * N[k * matrixSize + col];
		}
		P[row * matrixSize + col] = sum;
	}
}

// Function to perform matrix multiplication on CPU
void matMulCPU(float *M, float *N, float *P, int matrixSize) {
	for (int i = 0; i < matrixSize; i++) {
		for (int j = 0; j < matrixSize; j++) {
			float Pvalue = 0;
			for (int k = 0; k < matrixSize; k++) {
				Pvalue += M[j * matrixSize + k] * N[k * matrixSize + i];
			}
			P[j * matrixSize + i] = Pvalue;
		}
	}
}


int main()
{
	// Sizes of input matrices to test
	int sizeOfBlock = 16;
	int sizes[] = { 128, 256, 250, 1024, 2048 };
	int blockSizes[] = { 2,4,10,20,25 };
	//sizeOfBlock = blockSizes[4];
	printf("Block width of: %d\n", sizeOfBlock);

	// Loop over matrix sizes
	//for (int x = 0; x < 5; ++x) {
		for (int i = 0; i < 5; i++)
		{
			int size = sizes[i];
			
			printf("Matrix size is %d by %d\n\n", size, size);
			size_t hostSize = size * size * sizeof(float);

			float gpu_time1 = 0.0f;
			float gpu_time2 = 0.0f;

			// Allocate memory for input matrices on host
			float* h_M = (float*)malloc(hostSize);
			float* h_N = (float*)malloc(hostSize);
			float* h_C = (float*)malloc(hostSize);
			float* h_P = (float*)malloc(hostSize);


			srand(time(NULL));
			for (int i = 0; i < size * size; i++) {
				h_M[i] = (float)rand() / RAND_MAX;
				h_N[i] = (float)rand() / RAND_MAX;
			}

			// Allocate memory for input matrices on device
			float *d_M, *d_N, *d_C;
			hipMalloc(&d_M, hostSize);
			hipMalloc(&d_N, hostSize);
			hipMalloc(&d_C, hostSize);

			// Create events to measure time
			hipEvent_t start1, stop1, start2, stop2;
			hipEventCreate(&start1);
			hipEventCreate(&stop1);
			hipEventCreate(&start2);
			hipEventCreate(&stop2);

			// Copy input matrices from host to device and measure time
			//hipEventRecord(start1);
			hipMemcpy(d_M, h_M, hostSize, hipMemcpyHostToDevice);
			hipMemcpy(d_N, h_N, hostSize, hipMemcpyHostToDevice);
			// hipEventRecord(stop1);
			// hipEventSynchronize(stop1);
			// float transfer_time = 0;
			// hipEventElapsedTime(&transfer_time, start1, stop1);
			// printf("Matrix size %d x %d: Host to device transfer time = %f ms\n", size, size, transfer_time);

			int n_blocks = ceil(size / BLOCK_WIDTH);

			dim3 threadsPerBlock(sizeOfBlock, sizeOfBlock);
			dim3 numberOfBlocks(n_blocks, n_blocks);
			//intf("%d\n", ceil((size + threadsPerBlock.x - 1) / threadsPerBlock.x));
			//dim3 numberOfBlocks(ceil((size + threadsPerBlock.x - 1) / threadsPerBlock.x), ceil((size + threadsPerBlock.y - 1) / threadsPerBlock.y));


			// //Part 2 ---------------------------------------------------------------------
			hipEventRecord(start2, 0);
			matrixMultiplication << <numberOfBlocks, threadsPerBlock >> >(d_N, d_M, d_C, size);
			hipEventRecord(stop2, 0);
			hipEventSynchronize(stop2);
			hipEventElapsedTime(&gpu_time2, start2, stop2);
			hipMemcpy(h_C, d_C, hostSize, hipMemcpyDeviceToHost);
			printf("Normal Multiplication time: %0.2f\n", gpu_time2);



			//hipEventRecord(start1, 0);
			//matMulCPU(h_M, h_N, h_P, size);
			//hipEventRecord(stop1, 0);

			//hipEventElapsedTime(&gpu_time1, start1, stop1);
			//printf("Host Multiplication time: %0.2f\n\n", gpu_time1);

			//-----------------------------------------------------------------------------

			// Copy input matrices from device to host and measure time
			// hipEventRecord(start1);
			// hipMemcpy(h_M, d_M, hostSize, hipMemcpyDeviceToHost);
			// hipMemcpy(h_N, d_N, hostSize, hipMemcpyDeviceToHost);
			// hipEventRecord(stop1);
			// hipEventSynchronize(stop1);
			// transfer_time = 0;
			// hipEventElapsedTime(&transfer_time, start1, stop1);
			// printf("Matrix size %d x %d: Device to host transfer time = %f ms\n", size, size, transfer_time);

			// Free memory
			hipHostFree(h_M);
			hipHostFree(h_N);
			hipHostFree(h_C);
			hipHostFree(h_P);
			hipFree(d_M);
			hipFree(d_N);
			hipFree(d_C);
		}
	//}
	return 0;
}
