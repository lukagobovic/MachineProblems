//Luka Gobovic
//20215231
//MP3 Part 1
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <hiprand/hiprand.h>
#include <time.h>


int main()
{
	// Sizes of input matrices to test
	int sizes[] = { 125, 250, 500, 1000, 2000 };
	int num_sizes = sizeof(sizes) / sizeof(int);

	// Loop over matrix sizes
	for (int i = 0; i < num_sizes; i++)
	{
		int size = sizes[i];
		int num_elements = size * size;

		// Allocate memory for input matrices on host
		float *h_M, *h_N;
		hipHostMalloc(&h_M, num_elements * sizeof(float), hipHostMallocDefault);
		hipHostMalloc(&h_N, num_elements * sizeof(float), hipHostMallocDefault);

		srand(time(NULL));
		for (int i = 0; i < size * size; i++) {
			h_M[i] = (float)rand() / RAND_MAX;
			h_N[i] = (float)rand() / RAND_MAX;
		}

		// Allocate memory for input matrices on device
		float *d_M, *d_N;
		hipMalloc(&d_M, num_elements * sizeof(float));
		hipMalloc(&d_N, num_elements * sizeof(float));

		// Create events to measure time
		hipEvent_t start1, stop1, start2, stop2;
		hipEventCreate(&start1);
		hipEventCreate(&stop1);
		hipEventCreate(&start2);
		hipEventCreate(&stop2);

		// Copy input matrices from host to device and measure time
		hipEventRecord(start1);
		hipMemcpy(d_M, h_M, num_elements * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_N, h_N, num_elements * sizeof(float), hipMemcpyHostToDevice);
		hipEventRecord(stop1);
		hipEventSynchronize(stop1);
		float transfer_time = 0;
		hipEventElapsedTime(&transfer_time, start1, stop1);
		printf("Matrix size %d x %d: Host to device transfer time = %f ms\n", size, size, transfer_time);

		// Copy input matrices from device to host and measure time
		hipEventRecord(start1);
		hipMemcpy(h_M, d_M, num_elements * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(h_N, d_N, num_elements * sizeof(float), hipMemcpyDeviceToHost);
		hipEventRecord(stop1);
		hipEventSynchronize(stop1);
		transfer_time = 0;
		hipEventElapsedTime(&transfer_time, start1, stop1);
		printf("Matrix size %d x %d: Device to host transfer time = %f ms\n", size, size, transfer_time);

		// Free memory
		hipHostFree(h_M);
		hipHostFree(h_N);
		hipFree(d_M);
		hipFree(d_N);
	}

	return 0;
}
